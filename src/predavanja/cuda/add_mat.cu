#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define ROWS 1024*8
#define COLS 1024*8
#define MAT_SIZE ROWS * COLS
#define BLOCK_SIZE 16 // 2^4
// Host
float *h_ma;
float *h_mb;
float *h_mc;

// Device (GPU)
float *d_ma;
float *d_mb;
float *d_mc;


// Koda za GPU napravo
// __global__ (KERNEL) funkcija, se izvaja na napravi, zaganan iz gostitelja 
// __device__ funkcija, se izvaja na napravi, zagana na napravi

__global__ void add_mat(float *mata, float *matb, float *matc)
{
    // Index bloka * veliksot bloka 
    // + nit znotraj bloka
    // Dobimo globalni index vseh niti (1M)
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    matc[x * ROWS + y] = mata[x * ROWS + y]  + matb[x * ROWS + y];

}


//(const float *a, const float *b, const float *c, const in n)



// Koda za gostitelja (CPU)
int main(int argc, char *argv[])
{
    // Rezerviramo prostor na pomnilniku gostitelja
    h_ma = (float *)malloc(MAT_SIZE * sizeof(float));
    h_mb = (float *)malloc(MAT_SIZE * sizeof(float));
    h_mc = (float *)malloc(MAT_SIZE * sizeof(float));

    // Rezerviramo prostor na pomnilniku GPU
    hipMalloc(&d_va, MAT_SIZE * sizeof(float));
    hipMalloc(&d_vb, MAT_SIZE * sizeof(float));
    hipMalloc(&d_vc, MAT_SIZE * sizeof(float));

    // Vector initialization
    for (size_t i = 0; i < ROWS; i++) {
        for (size_t j = 0; i < COLS; j++)
        {
            h_ma[i * ROWS + j] = 9.0f;
            h_mb[i * ROWS + j] = 4.0f;
        }
    }

    // hipMemcpy
    // Naslov ponor podatkov (destination)
    // Naslov izvora podatkov (source)
    // Stevilo B za prenos
    // Smer prenosa (H -> D, D -> H)

    // Prenos podatkov iz pomnilnika gostitelja v pomnilnik naprav3
    cudaMemecpy((void *)d_ma,
                (void *)h_ma,
                MAT_SIZE * sizeof(float),
                hipMemcpyHostToDevice);

    cudaMemecpy((void *)d_mb,
                (void *)h_mb,
                MAT_SIZE * sizeof(float),
                hipMemcpyHostToDevice);

    // Zazeni kernel na napravi (GPU)
    // Niti v 1 bloku --> tvorjenje snopov
    // Skupaj blok 256 (2^8)
    // rows: 2^4, cols: 2^4 (16x16)
    // rows: 2^5, cols: 2^3 (32x8)
    dim3 threadsInBlock[BLOCK_SIZE, BLOCK_SIZE, 1];
    // St blokov 
    // Y (rows): (2^20 / 2^4 = 2^16) 64K (1024 * 1024 elements)
    // X (cols): (2^10 / 2^4 = 2^6) 64 (1024 elements)
    dim3 numOfBlocks[ROWS/BLOCK_SIZE, COLS/BLOCK_SIZE, 1];

    add_mat<<<numOfBlocks, threadsInBlock>>>>(d_ma, d_mb, d_mc);

    // Prevajanje programa 
    // srun --partition=gpu nvcc dotprod.cu -o dtoprod
    // srun --partition=gpu --ntask=1 --gpus=1 --mem-per-cpu=1600MB? 

    // Prenos rezultate iz naprave v gostitelja
    cudaMemecpy((void *)h_mc,
                (void *)d_mc,
                MAT_SIZE * sizeof(float),
                hipMemcpyDeviceToHost);


    // Print result (should be 6)
    printf("Element: %d", h_mc[567 * ROWS + 120]);

    // Sprostimo prostor gostitelja
    free(h_ma);
    free(h_mb);
    free(h_mc);

    // Sprostimo prostor na naprvi (GPU)
    hipFree(d_ma);
    hipFree(d_mb);
    hipFree(d_mc);

    return 0;
}

