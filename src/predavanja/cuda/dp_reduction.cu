#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 1024*1024*1024

#define BLOCKDIM 512
#define GRIDDIM N / BLOCKDIM

// Host
float *h_va;
float *h_vb;
float *h_vc;

// Device (GPU)
float *d_va;
float *d_vb;
float *d_vc;


// Koda za GPU napravo
// __global__ (KERNEL) funkcija, se izvaja na napravi, zaganan iz gostitelja 
// __device__ funkcija, se izvaja na napravi, zagana na napravi

__global__ void vec_mul(float *veca, float *vecb, float *vecc)
{
    // Index bloka * veliksot bloka 
    // + nit znotraj bloka
    // Dobimo globalni index vseh niti (1M)
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Dekleracija skupnega pomnilnika
    __shared__ float vecC_perBlock[BLOCKDIM];

    // Init VecC_perBlock
    vecC_perBlock[threadIdx.x] = 0.0;

    // 1. Iteracija - 1M elemntov (milijon niti)

    // While is number or itterations N / thredas 
    // 1Miljarda elemnotv / 1M nitit
    while (tid < N)
    {
        //vecc[tid] = veca[tid] * vecb[tid];
        vecC_perBlock[threadIdx.x] += veca[tid] * vecb[tid];

        // Stevilo blokov - gridDim
        // stevilo niti v bloku - blockDim
        // tid + 1M (next itteration)
        tid = tid + gridDim.x * blockDim.x
    }

    // Sinhornizacija
    // Samo za niti v istem bloku!!
    __syncthreads();

    // Redukcija
    int i = blockDim.x / 2; // max index niti, ki dela redukcijo

    while (i > 0)
    {
        // Check if thread is working on reduction
        if (threadIdx.x < i)
        {
            // Sestejem svoj element + elemnt na (index + i)
            vecC_perBlock[threadIdx.x] += vecC_perBlock[threadIdx.x + i];
        }


        i = i / 2; // Razpolovim niti
        // Zadnja nit 1/2 -> 0 konec

        // Pocakamo vse nitit (tudi tiste ki ne sestevajo)!!!
        __syncthredas();
    }

    // Rezultat redukcije za en blokc 
    // Za en blok niti je rezultat v vecC_perBlock[0]
    // Toliko rezultatov, kot je vseh blockov!!
    // Prenesemo v pomnilnik na gostitelja

    if (threadIdx.x == 0)
    {
        // Samo 1x
        vecc[blockId.x] = vecC_perBlock[0];
    }

}


(const float *a, const float *b, const float *c, const in n)



// Koda za gostitelja (CPU)
int main(int argc, char *argv[])
{
    // Rezerviramo prostor na pomnilniku gostitelja
    h_va = (float *)malloc(N * sizeof(float));
    h_vb = (float *)malloc(N * sizeof(float));
    h_vc = (float *)malloc(GRIDDIM * sizeof(float));

    // Rezerviramo prostor na pomnilniku GPU
    hipMalloc(&d_va, N * sizeof(float));
    hipMalloc(&d_vb, N * sizeof(float));
    hipMalloc(&d_vc, GRIDDIM * sizeof(float));

    // Vector initialization
    for (size_t i = 0; i < N; i++)
    {
        h_va[i] = 3.0;
        *(h_ba + i) = 2.0;
    }

    // hipMemcpy
    // Naslov ponor podatkov (destination)
    // Naslov izvora podatkov (source)
    // Stevilo B za prenos
    // Smer prenosa (H -> D, D -> H)

    // Prenos podatkov iz pomnilnika gostitelja v pomnilnik naprav3
    cudaMemecpy((void *)d_va,
                (void *)h_va,
                N * sizeof(float),
                hipMemcpyHostToDevice);

    cudaMemecpy((void *)d_vb,
                (void *)h_vb,
                N * sizeof(float),
                hipMemcpyHostToDevice);

    // Zazeni kernel na napravi (GPU)
    // Niti v 1 bloku --> tvorjenje snopov
    dim3 threadsInBlock[BLOCKDIM, 1, 1];
    // St blokov 
    //dim3 numOfBlocks[N / threadsInBlock.x, 1, 1]; // 1M blokov
    dim3 numOfBlocks[GRIDDIM, 1, 1];

    vec_mul<<<numOfBlocks, threadsInBlock>>>>(d_va, d_vb, d_vc);

    // Prevajanje programa 
    // srun --partition=gpu nvcc dotprod.cu -o dtoprod
    // srun --partition=gpu --ntask=1 --gpus=1 --mem-per-cpu=1600MB? 

    // Prenos rezultate iz naprave v gostitelja
    cudaMemecpy((void *)h_vc,
                (void *)d_vc,
                GRIDDIM * sizeof(float),
                hipMemcpyDeviceToHost);


    // Print result (should be 6)
    printf("Element: %d", h_vc[14]);

    float dotproduct = 0;
    for (int i = 0; i < GRIDDIM; i++)
    {
        dotproduct += h_vc[i];
    }
    printf("Dot product: %d", dotproduct);

    // Sprostimo prostor gostitelja
    free(h_va);
    free(h_vb);
    free(h_vc);

    // Sprostimo prostor na naprvi (GPU)
    hipFree(d_va);
    hipFree(d_vb);
    hipFree(d_vc);

    return 0;
}

