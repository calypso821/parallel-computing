#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define ROWS 1024*8
#define COLS ROWS // Cols = Rows

#define BLOCKDIM 32 // 2^5
#define N ROWS
// Host
float *h_ma;
float *h_mb;
float *h_mc;

// Device (GPU)
float *d_ma;
float *d_mb;
float *d_mc;


// Koda za GPU napravo
// __global__ (KERNEL) funkcija, se izvaja na napravi, zaganan iz gostitelja 
// __device__ funkcija, se izvaja na napravi, zagana na napravi

__global__ void mul_mat_naive(float *mata, float *matb, float *matc)
{
    // Index bloka * veliksot bloka 
    // + nit znotraj bloka
    // Dobimo globalni index vseh niti (1M)
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Stevilo niti = velikost matrike C (ROWS * COLS)
    // Vsaka nit racuna 1 element matrike C

    matc[row * ROWS + col] = 0.0;

    for (size_t i = 0; i < N; i++ )
    {
        // Memory coalessing PROBLEM!!
        // Sosednji element dosotopa do istih elemntov ko prejsnji (row, col)
        // V celotni vrstici / stolpcu dostopamo do enakih elemntov

        // matC[row, col] = matA[row, i:0..N] * matB[i:0..N, col]
        // Premik po vrstici --> mata[row * ROWS + i]
        // Premik po stolpcu --> matb[i * ROWS + col]
        matc[row * ROWS + col] += mata[row * ROWS + i] * matb[i * ROWS + col];
    }
}


__global__ void mul_mat_tiles(float *mata, float *matb, float *matc)
{
    // Index bloka * veliksot bloka 
    // + nit znotraj bloka
    // Dobimo globalni index vseh niti (1M)
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    int local_row = threadIdx.y;
    int local_col = threadIdx.x;

    float c = 0.0;

    // Prostor za plscici v shread memory
    __shared__ float tileA[BLOCKDIM][BLOCKDIM];
    __shared__ float tileB[BLOCKDIM][BLOCKDIM];

    // Po vseh ploscicah (tilih) po matriki A in B, ki jih potrebujemo za izracun C
    // Izbiram in nalagam jih po parih

    for (int tile = 0; tile < (N / BLOCKDIM); tile++)
    {
        // preberimo poscici A in B iz globalnega pomnilnika
        // Vsaka nit v bloku prenese 1 element
        // Lokalni ploscicic 

        // * BLOCKDIM (linearizacija)?? 
        // mata[blockIdx.y * blockDim.y + local_row][tile * blockDim.x + local_col] NOT OK??? 
        // mata in matb ni matrika?? Just array?? 

        tileA[local_row][local_col] = mata[(blockIdx.y * blockDim.y + local_row) * BLOCKDIM + 
                                            (tile * blockDim.x + local_col)]
        tileB[local_row][local_col] = matb[(tile * blockDim.y + local_row) * BLOCKDIM + 
                                            (blockIdx.x * blockDim.x + local_col)]


        // Pocakaj, da vse niti prenesejo svoje elemente
        __syncthreads();

        // Zmnozi ploscici in rezultat pisi v plosico C
        for (int i = 0; i < BLOCKDIM; i++)
        {
            c += tileA[local_row][i] * tileB[i][local_col];
        }

        // Pocakaj, pred ponovnim nalaganjem novih ploscic
        __syncthreads();
    }

    // Prenesi element c nazaj v globalni pomnilnik v matriko C
    // What is rowN???
    // ROWS?? or
    matc[rowN, col] = c;

}

//(const float *a, const float *b, const float *c, const in n)



// Koda za gostitelja (CPU)
int main(int argc, char *argv[])
{
    // Rezerviramo prostor na pomnilniku gostitelja
    h_ma = (float *)malloc(MAT_SIZE * sizeof(float));
    h_mb = (float *)malloc(MAT_SIZE * sizeof(float));
    h_mc = (float *)malloc(MAT_SIZE * sizeof(float));

    // Rezerviramo prostor na pomnilniku GPU
    hipMalloc(&d_va, MAT_SIZE * sizeof(float));
    hipMalloc(&d_vb, MAT_SIZE * sizeof(float));
    hipMalloc(&d_vc, MAT_SIZE * sizeof(float));

    // Vector initialization
    for (size_t i = 0; i < ROWS; i++) {
        for (size_t j = 0; i < COLS; j++)
        {
            h_ma[i * ROWS + j] = 1.0f;
            h_mb[i * ROWS + j] = 2.0f;
        }
    }

    // hipMemcpy
    // Naslov ponor podatkov (destination)
    // Naslov izvora podatkov (source)
    // Stevilo B za prenos
    // Smer prenosa (H -> D, D -> H)

    // Prenos podatkov iz pomnilnika gostitelja v pomnilnik naprav3
    cudaMemecpy((void *)d_ma,
                (void *)h_ma,
                MAT_SIZE * sizeof(float),
                hipMemcpyHostToDevice);

    cudaMemecpy((void *)d_mb,
                (void *)h_mb,
                MAT_SIZE * sizeof(float),
                hipMemcpyHostToDevice);

    // Zazeni kernel na napravi (GPU)
    // Niti v 1 bloku --> tvorjenje snopov
    // Skupaj blok 256 (2^8)
    // rows: 2^4, cols: 2^4 (16x16)
    // rows: 2^5, cols: 2^3 (32x8)
    dim3 threadsInBlock[BLOCKDIM, BLOCKDIM, 1];
    // St blokov 
    // Y (rows): (2^20 / 2^4 = 2^16) 64K (1024 * 1024 elements)
    // X (cols): (2^10 / 2^4 = 2^6) 64 (1024 elements)
    dim3 numOfBlocks[ROWS/BLOCKDIM, COLS/BLOCKDIM, 1];

    // CUDA events for measuring times
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);


    // Start recording
    cudaEventRecorde(start);

    add_mat<<<numOfBlocks, threadsInBlock>>>>(d_ma, d_mb, d_mc);

    // Stop recording
    cudaEventRecorde(end);
    hipEventSynchronize(end);

    float miliseconds = 0.0;
    hipEventElapsedTime(&miliseconds, start, end);
    printf("Kernel execution time: %0.3f miliseconds\n", miliseconds);

    // Prevajanje programa 
    // srun --partition=gpu nvcc dotprod.cu -o dtoprod
    // srun --partition=gpu --ntask=1 --gpus=1 --mem-per-cpu=1600MB? 

    // Prenos rezultate iz naprave v gostitelja
    cudaMemecpy((void *)h_mc,
                (void *)d_mc,
                MAT_SIZE * sizeof(float),
                hipMemcpyDeviceToHost);


    // Print result (should be 6)
    printf("Element: %d", h_mc[567 * ROWS + 120]);
    // Delni produkt bedno 2 (1 * 2)
    // Dimenzija = 1024 * 8 
    // 2 * 8K --> 16K
    // Element should be ~ 16k

    // Sprostimo prostor gostitelja
    free(h_ma);
    free(h_mb);
    free(h_mc);

    // Sprostimo prostor na naprvi (GPU)
    hipFree(d_ma);
    hipFree(d_mb);
    hipFree(d_mc);

    return 0;
}

